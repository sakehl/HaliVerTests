#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
//Kernel
////////////////////////////////////////////////////////////////////////////////
/*@ 
context_everywhere array != NULL;
context_everywhere output != NULL;
context_everywhere size == 1024;
context_everywhere opencl_gcount == 1;
context_everywhere opencl_gsize == size;
//context \pointer_index(array, \ltid, write); This passes
//context \pointer_index(array, \ltid, 1); This gives an error:  unspecified error z3.interaction.failed java.lang.ArrayIndexOutOfBoundsException: Index 1 out of bounds for length 1
//context 2*\ltid < size ==> \pointer_index(array, 2*\ltid, write); This gives  CallPreCondition:AssertionFalse
//context 2*\ltid+1 < size ==> \pointer_index(array, 2*\ltid+1, write); This gives  CallPreCondition:AssertionFalse
requires \ltid > 0 ==> \pointer_index(array, \ltid - 1, 1\2); 
//requires \pointer_index(output, \ltid, 1\2);
//context \ltid < (size+2-1)/2 ==> \pointer_index(array, 2*\ltid, write); //This gives  CallPreCondition:AssertionFalse
//context \ltid < (size-1+2-1)/2 ==> \pointer_index(array, 2*\ltid+1, write); //This gives  CallPreCondition:AssertionFalse
@*/
__global__ void CUDA_Kernel(int* array, int* output, int size)
{
  
  int tid = threadIdx.x;
  //@ assert tid == \ltid;
  
}

////////////////////////////////////////////////////////////////////////////////
// CUDA Functions
////////////////////////////////////////////////////////////////////////////////
//@ ensures \pointer(\result, N, write);
int *vercorsMallocInt(int N);
void vercorsFreeInt(int *ar);
//@ ensures \pointer(\result, N, write);
int *vercorsCudaMallocInt(int N);
void vercorsCudaFreeInt(int *addr);
//@ context \pointer(src, N, read) ** \pointer(tgt, N, write);
//@ ensures (\forall int i; i >= 0 && i < N; src[i] == tgt[i]);
void vercorsCudaMemcpyInt(int *tgt, int *src, int N, int direction); 

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////

int CUDA_Host_Blelloch( int argc, char** argv){
  
  int size = 1024; // size of the input 
  
  int* host_output = vercorsMallocInt(size); 
  
  int* device_array;
  device_array = vercorsCudaMallocInt(size);
  vercorsCudaMemcpyInt(device_array, host_output, size, hipMemcpyHostToDevice) ;
  
  int* device_output;
  device_output = vercorsCudaMallocInt(size);
  
  //setup execution parameters
  int num_of_blocks = 1;
  int num_of_threads_per_block = size;
  
  CUDA_Kernel<<< /*grid*/num_of_blocks, /*threads*/num_of_threads_per_block/*, 0*/ >>>(device_array, device_output, size);
  
  //vercorsCudaMemcpyInt(host_output, device_output, size, cudaMemcpyDeviceToHost) ;
  
  // cleanup memory
  vercorsFreeInt(host_output);
  vercorsCudaFreeInt(device_output);
  
}





