#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
//Kernel
////////////////////////////////////////////////////////////////////////////////
/*@ 
context_everywhere array != NULL;
context_everywhere output != NULL;
context_everywhere size == 1024;
context_everywhere opencl_gcount == 1;
context_everywhere opencl_gsize == size;
//context \pointer_index(array, \ltid, write); This passes
//context \pointer_index(array, \ltid, 1); This gives an error:  unspecified error z3.interaction.failed java.lang.ArrayIndexOutOfBoundsException: Index 1 out of bounds for length 1
//context 2*\ltid < size ==> \pointer_index(array, 2*\ltid, write); This gives  CallPreCondition:AssertionFalse
//context 2*\ltid+1 < size ==> \pointer_index(array, 2*\ltid+1, write); This gives  CallPreCondition:AssertionFalse
requires \ltid > 0 ==> \pointer_index(array, \ltid - 1, 1\2); 
//requires \pointer_index(output, \ltid, 1\2);
//context \ltid < (size+2-1)/2 ==> \pointer_index(array, 2*\ltid, write); //This gives  CallPreCondition:AssertionFalse
//context \ltid < (size-1+2-1)/2 ==> \pointer_index(array, 2*\ltid+1, write); //This gives  CallPreCondition:AssertionFalse
@*/
__global__ void CUDA_Kernel(int* array, int* output, int size)
{
  
  int tid = threadIdx.x;
  //@ assert tid == \ltid;
  
}




////////////////////////////////////////////////////////////////////////////////
// CUDA Functions
////////////////////////////////////////////////////////////////////////////////
//@ ensures \pointer(\result, N, write);
int *vercorsMallocInt(int N);
void vercorsFreeInt(int *ar);
//@ ensures \pointer(\result, N, write);
int *vercorsCudaMallocInt(int N);
void vercorsCudaFreeInt(int *addr);
//@ context \pointer(src, N, read) ** \pointer(tgt, N, write);
//@ ensures (\forall int i; i >= 0 && i < N; src[i] == tgt[i]);
void vercorsCudaMemcpyInt(int *tgt, int *src, int N, int direction); 




